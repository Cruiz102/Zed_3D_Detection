#include "hip/hip_runtime.h"
__global__ void backProject2Dto3D(float *d_x, float *d_y, float *d_depth, float *d_K, float *d_R, float *d_t, float *d_output, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        float fx = d_K[0];
        float fy = d_K[4];
        float cx = d_K[2];
        float cy = d_K[5];

        float X_cam = (d_x[i] - cx) * d_depth[i] / fx;
        float Y_cam = (d_y[i] - cy) * d_depth[i] / fy;
        float Z_cam = d_depth[i];

        if (d_R != NULL && d_t != NULL)
        {
            // Assuming R is a 3x3 matrix and t is a 3x1 vector
            float P_cam[3] = {X_cam, Y_cam, Z_cam};
            for (int j = 0; j < 3; j++)
            {
                d_output[3 * i + j] = 0;
                for (int k = 0; k < 3; k++)
                {
                    d_output[3 * i + j] += d_R[j * 3 + k] * (P_cam[k] - d_t[k]);
                }
            }
        }
        else
        {
            d_output[3 * i] = X_cam;
            d_output[3 * i + 1] = Y_cam;
            d_output[3 * i + 2] = Z_cam;
        }
    }
}



// Function to run the kernel
void runBackProject2Dto3D(float *h_output, int n) {
    float *d_output;
    hipMalloc(&d_output, n * sizeof(float));

    // Define the execution configuration
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch the kernel
    backProject2Dto3D<<<numBlocks, blockSize>>>(d_output, n);

    // Copy the result back to the host
    hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_output);
}
